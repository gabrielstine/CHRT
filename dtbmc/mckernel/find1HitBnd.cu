
#include <hip/hip_runtime.h>
//
// GPU PTX kernel to find the index of 1st up or lower bound hitting
//

// Copyright 2014 Jian Wang


__global__ void find1HitBnd(const bool* upBnd, double* iup,
                            const bool* loBnd, double* ilo,
                            const int ntime, const int ntrials)
{
    int thisThread = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thisThread < ntrials) {
        int m = thisThread * ntime;
        for (int n = 0; n < ntime; ++n) {
            if (upBnd[m]) {
                iup[m] = 1.0;
                break;
            }
            else if (loBnd[m]) {
                ilo[m] = 1.0;
                break;
            }
            ++m;
        }
    }
}