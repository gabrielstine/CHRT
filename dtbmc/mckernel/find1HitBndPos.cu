
#include <hip/hip_runtime.h>
//
// GPU PTX kernel to find the index of 1st up or lower bound hitting
// 
//

// Copyright 2014 Jian Wang


__global__ void find1HitBndPos(const bool* upBnd, double* iup,
                            const bool* loBnd, double* ilo,
                            const int ntime, const int ntrials,
                            const double* dft, double* ipos)
{
    int thisThread = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thisThread < ntrials) {
        int m = thisThread * ntime;
        for (int n = 0; n < ntime; ++n) {
        	if (dft[m] > 0.0) {        		
        	    ipos[m] = 1.0;        	    
        	}
        	
            if (upBnd[m]) {
                iup[m] = 1.0;
                break;
            }
            else if (loBnd[m]) {
                ilo[m] = 1.0;
                break;
            }
                        
            ++m;
        }
    }
}