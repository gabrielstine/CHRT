
#include <hip/hip_runtime.h>
//
// CUDA PTX kernel to find the first index of hitting upper bound in DTB 
// 2D Monte Carlo simulation.
//

// Copyright 2015 Jian Wang

__global__ void withoutPos(const double* randpool, 
		const double* bup, const double* blo, double* iup, double* ilo,                                                        
		const int ntime, const int ntrials, 
		const double y0, const double mu, const double cov)
{
	int thisThread = blockIdx.x * blockDim.x + threadIdx.x;

	if (thisThread < ntrials) {                
		double rup = y0, rlo = y0; // Upper & lower competing races. 
		double thisBUp, thisBLow; // Upper & lower bound height at this time.

		int m = thisThread * ntime * 3.0; // Index of rand number.
		int k = thisThread * ntime; // Index of boundary.

		for (int n = 0; n < ntime-1; ++n) {
			rup += ((1-abs(cov))*randpool[m] + abs(cov)*randpool[m+1]) + mu;
			rlo += ((1-abs(cov))*randpool[m+2] + cov*randpool[m+1])    - mu;

			thisBUp = bup[n+1];
			thisBLow = blo[n+1];

			// Check whether hitting lower boundary.
			if (rup < thisBLow) { 
				rup = thisBLow;
			}

			if (rlo < thisBLow) {
				rlo = thisBLow;
			}

			// Check whether hitting upper boundary.
			if (rup > thisBUp) {
				iup[k+1] = 1.0;
				break;
			}
			else if (rlo > thisBUp) {
				ilo[k+1] = 1.0;
				break;
			}

			m += 3;
			k++;
		}
	}
}


//
// CUDA PTX kernel to find the first index of hitting upper bound with pos in DTB 
// 2D Monte Carlo simulation.
//

// Copyright 2015 Jian Wang

__global__ void withPos(const double* randpool, 
		const double* bup, const double* blo, double* iup, double* ilo,                                                        
		const int ntime, const int ntrials, 
		const double y0, const double mu, const double cov, double* pos)
{
	int thisThread = blockIdx.x * blockDim.x + threadIdx.x;

	if (thisThread < ntrials) {                
		double rup = y0, rlo = y0; // Upper & lower competing races. 
		double thisBUp, thisBLow; // Upper & lower bound height at this time.

		int m = thisThread * ntime * 3.0; // Index of rand number.
		int k = thisThread * ntime; // Index of boundary.

		for (int n = 0; n < ntime-1; ++n) {
			rup += ((1-abs(cov))*randpool[m] + abs(cov)*randpool[m+1]) + mu;
			rlo += ((1-abs(cov))*randpool[m+2] + cov*randpool[m+1])    - mu;

			thisBUp = bup[n+1];
			thisBLow = blo[n+1];

			// Check whether hitting lower boundary.
			if (rup < thisBLow) { 
				rup = thisBLow;
			}

			if (rlo < thisBLow) {
				rlo = thisBLow;
			}
			
			// Check positive non-absorptive possibility.
			if (rup > rlo) {
				pos[k+1] = 1.0;
			}				

			// Check whether hitting upper boundary.
			if (rup > thisBUp) {
				iup[k+1] = 1.0;
				break;
			}
			else if (rlo > thisBUp) {
				ilo[k+1] = 1.0;
				break;
			}

			m += 3;
			k++;
		}
	}
}



