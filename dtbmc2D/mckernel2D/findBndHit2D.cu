#include "hip/hip_runtime.h"
//
// GPU PTX kernel to find the index of 1st up or lower bound hitting.
//

// Copyright 2014 Jian Wang

__global__ void withoutPos(const double* randpool, 
		const double* bup, const double* blo, double* iup, double* ilo,                                                        
		const int ntime, const int ntrials, 
		const double y0, const double mu, const double cov)
{
    int thisThread = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thisThread < ntrials) {                
        double rup = y0, rlo = y0; // up & lo 2 races. 
        int m = thisThread * (ntime * 3.0); // 3 rand number per time step.
        
        for (int n = 0; n < ntime; ++n) {
        		rup +=  ((1-abs(cov))*randpool[m] + abs(cov)*randpool[m+1]) + mu;
        		rlo  +=  ((1-abs(cov))*randpool[m+2]) + cov*randpool[m+1])   + mu;
        	
        		thisBUp = bup[n+1];
        		thisBLow = blo[n+1];

        		// Check hitting lower boundary.
        		if (rup < thisBLow) 
        			rup = thisBLow;
        		
        		if (rlo < thisBLow)
        			rlo = thisBLow;
        		
        		// Check hitting up boundary.
            if ((rup > rlo) && (rup > thisBUp) {
                iup[n+1] = 1.0;
                break;
            }
            else if ((rup < rlo) && (rlo > thisBUp)) {
                ilo[n+1] = 1.0;
                break;
            }
                                    
            m += 3;
        }
    }
}


//
// GPU PTX kernel to find the index of 1st up or lower bound hitting
// and the possibility of in positive half axis.
// 

// Copyright 2014 Jian Wang

//__global__ void withPos(const bool* upBnd, double* iup,		
//                        const bool* loBnd, double* ilo,
//                        const int ntime, const int ntrials,
//                        const double* dft, double* ipos)
//{
//    int thisThread = blockIdx.x * blockDim.x + threadIdx.x;
//    
//    if (thisThread < ntrials) {
//        int m = thisThread * ntime;
//        for (int n = 0; n < ntime; ++n) {
//        	if (dft[m] > 0.0) {        		
//        	    ipos[m] = 1.0;        	    
//        	}
//        	
//            if (upBnd[m]) {
//                iup[m] = 1.0;
//                break;
//            }
//            else if (loBnd[m]) {
//                ilo[m] = 1.0;
//                break;
//            }
//                        
//            ++m;
//        }
//    }
//}


