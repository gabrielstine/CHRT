
#include <hip/hip_runtime.h>
//
// CUDA PTX kernel to find the first index of hitting upper bound in DTB 
// 2D Monte Carlo simulation.
//

// Copyright 2015 Jian Wang

__global__ void withoutPos(const double* randpool, 
		const double* bup, const double* blo, double* iup, double* ilo,                                                        
		const int ntime, const int ntrials, 
		const double y0, const double mu, const double cov)
{
	int thisThread = blockIdx.x * blockDim.x + threadIdx.x;

	if (thisThread < ntrials) {                
		double rup = y0, rlo = y0; // Upper & lower competing races. 
		double thisBUp, thisBLow; // Upper & lower bound height at this time.

		int m = thisThread * ntime * 3.0; // Index of rand number.
		int k = thisThread * ntime; // Index of boundary.

		for (int n = 0; n < ntime-1; ++n) {
			rup += ((1-abs(cov))*randpool[m] + abs(cov)*randpool[m+1]) + mu;
			rlo += ((1-abs(cov))*randpool[m+2] + cov*randpool[m+1])    - mu;

			thisBUp = bup[n+1];
			thisBLow = blo[n+1];

			// Check whether hitting lower boundary.
			if (rup < thisBLow) { 
				rup = thisBLow;
			}

			if (rlo < thisBLow) {
				rlo = thisBLow;
			}

			// Check whether hitting upper boundary.
			if (rup > thisBUp) {
				iup[k+1] = 1.0;
				break;
			}
			else if (rlo > thisBUp) {
				ilo[k+1] = 1.0;
				break;
			}

			m += 3;
			k++;
		}
	}
}


//
// GPU PTX kernel to find the index of 1st up or lower bound hitting
// and the possibility of in positive half axis.
// 

// Copyright 2014 Jian Wang

//__global__ void withPos(const bool* upBnd, double* iup,		
//                        const bool* loBnd, double* ilo,
//                        const int ntime, const int ntrials,
//                        const double* dft, double* ipos)
//{
//    int thisThread = blockIdx.x * blockDim.x + threadIdx.x;
//    
//    if (thisThread < ntrials) {
//        int m = thisThread * ntime;
//        for (int n = 0; n < ntime; ++n) {
//        	if (dft[m] > 0.0) {        		
//        	    ipos[m] = 1.0;        	    
//        	}
//        	
//            if (upBnd[m]) {
//                iup[m] = 1.0;
//                break;
//            }
//            else if (loBnd[m]) {
//                ilo[m] = 1.0;
//                break;
//            }
//                        
//            ++m;
//        }
//    }
//}

